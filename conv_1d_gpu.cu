#include "hip/hip_runtime.h"
/*  Serial version of conv_1d for minibatch with variable length instances.
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <limits.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define MIN(a,b) ((a<b)?a:b)
#define MAX(a,b) ((a>b)?a:b)

#define DEBUG 1

typedef struct WORDVECS{
    float* w;
    int dim;
    long* lens;
    int b_size;
}WORDVECS;

typedef struct KERNS{
    float* k;
    int num;
    int width;
    int height;
}KERNS;

typedef struct OUTPUTS{
    float* out;
    int dim;
    long* lens;
    int b_size;
}OUTPUTS;


void read_sentence_lens(const char* file_path, int* sent_lens, int n_sents){
    /*
     Reads sentence lengths from Trip Advisor Dataset. Assumes that sentence
     lengths are non-zero.
     */
    int len=0;
    FILE *fp = fopen(file_path, "r");
    if (fp == NULL) {
        fprintf(stderr, "Can't open input file %s!\n", file_path);
        exit(1);
    }
    int fret = 1;
    for (int i=0; (i<n_sents && fret != EOF); i++) {
        fret = fscanf(fp, "%d\n", &len);
        if (len > 125) {
            len = 125;
        }
        sent_lens[i] = len;
    }
    fclose(fp);
}

void init_lens(long* lens, int* sent_lens, int batch_size, int batch){
    lens[0] = sent_lens[batch*batch_size];
    for (int i=1; i < batch_size; i++) {
        lens[i] = lens[i-1] + sent_lens[batch*batch_size + i];
    }
}

void init_wordvecs(float* wordvecs, int dim, int total_words){
    /*
     Initilizes word vectors. i.e input for convolution
     */
    for (int i=0; i < total_words; i++) {
        for (int d=0; d < dim; d++) {
            wordvecs[i*dim+d] = 1.;
        }
    }
}

void init_kerns(float* kerns, int n_kerns, int kern_w, int kern_h){
    /*
     Initilizes kernels.
     */
    for (int i=0; i < (n_kerns*kern_w*kern_h); i++) {
        kerns[i] = rand()/(float)INT_MAX;
    }
}

void init_out_lens(long** out_lens, long* lens, int b_size, int kern_w){
    (*out_lens)[0] = (lens[0] + kern_w - 1);
    for (int i=1; i < b_size; i++) {
        (*out_lens)[i] = (*out_lens)[i-1] + (lens[i] - lens[i-1]) + kern_w - 1;
    }
}


void print_mat(float* mat, int width,int height){
    /*
     Printing the matrix for verification.
     */
    printf("np.array([");
    for (int i=0; i<width; i++) {
        printf("[");
        for (int j=0; j<height; j++) {
            if (j == height-1) {
                printf(" %.4f", mat[i*height+j]);
            } else {
                printf(" %.4f,", mat[i*height+j]);
            }
        }
        if (i == width-1) {
            printf("]");
        } else {
            printf("],\n");
        }
    }
    printf("])\n");
}


void conv1d(WORDVECS wordvec, KERNS kerns, OUTPUTS output){
    /*
     Performs 1d convolution on CPU for each mini-batch at a time.
     */
    long len, out_len;
    float* wv;
    float* out;
    int dim = wordvec.dim, out_dim=kerns.num;
    for (int inst=0; inst < wordvec.b_size; inst++) {
        if (inst == 0) {
            len = wordvec.lens[inst];
            out_len = output.lens[inst];
            wv = &wordvec.w[dim*0];
            out = &output.out[out_dim*0];
        } else {
            len = wordvec.lens[inst] - wordvec.lens[inst-1];
            out_len = output.lens[inst] - output.lens[inst-1];
            wv = &wordvec.w[dim*wordvec.lens[inst-1]];
            out = &output.out[out_dim*output.lens[inst-1]];
        }
        for (int i=0; i < out_len; i++) {
            for (int k=0; k < kerns.num; k++) {
                float s = 0.;
                for (int j = MAX(0, i-kerns.width+1); j <= MIN(i, len-1); j++) {
                    int k_sub=(kerns.width-1-i+j);
                    for (int d=0; d<dim; d++) {
                        s += (wv[j*dim+d] * kerns.k[k*kerns.width*kerns.height + k_sub*kerns.height + d]);
                    }
                }
                out[i*kerns.num+k] += s;
            }
        }
    }
}

__global__
void conv1d_kernel(WORDVECS wordvec, KERNS kerns, OUTPUTS output){
    /*
     Performs 1d convolution on CPU for each mini-batch at a time.
     */
    int tIdx = threadIdx.x;
    int bIdx = blockIdx.x;
    
    long len, out_len;
    float* wv;
    float* out;
    int dim = wordvec.dim, out_dim=kerns.num;
    
    assert(blockDim.x == dim);
    
    extern __shared__ float s[];
    
    if (bIdx == 0) {
        len = wordvec.lens[bIdx];
        out_len = output.lens[bIdx];
        wv = &wordvec.w[dim*0];
        out = &output.out[out_dim*0];
    } else {
        len = wordvec.lens[bIdx] - wordvec.lens[bIdx-1];
        out_len = output.lens[bIdx] - output.lens[bIdx-1];
        wv = &wordvec.w[dim*wordvec.lens[bIdx-1]];
        out = &output.out[out_dim*output.lens[bIdx-1]];
    }
    __syncthreads();
    
    for (int i=0; i < out_len; i++) {
        for (int k=0; k < kerns.num; k++) {
            s[tIdx] = 0.;
            for (int j = MAX(0, i-kerns.width+1); j <= MIN(i, len-1); j++) {
                int k_sub=(kerns.width-1-i+j);
                s[tIdx] += (wv[j*dim+tIdx] * kerns.k[k*kerns.width*kerns.height + k_sub*kerns.height + tIdx]);
            }
            atomicAdd(&out[i*kerns.num+k], s[tIdx]);
            __syncthreads();
        }
    }
}


int main(int argc, char* argv[]){

    if (argc != 7) {
        printf("USAGE: ./conv_1d.o <n_batches> <batch_size> <dim> <kern_w> <n_kerns> <device_id>");
        exit(1);
    }
    
    //Initilizing random numbers
    srand(20);
    
    KERNS kerns;
    
    //Parsing commandline args and initialize structs
    int n_batches = atoi(argv[1]);
    int batch_size = atoi(argv[2]);
    int dim = atoi(argv[3]);
    kerns.height = dim;
    kerns.width = atoi(argv[4]);
    kerns.num = atoi(argv[5]);
    int device_id = atoi(argv[6]);
    printf("n_batches=%d, batch_size=%d, dim=%d, kern_w=%d, kern_h=%d, n_kerns=%d\n", n_batches, batch_size, dim, kerns.width, kerns.height, kerns.num);
    
    // Read mini-batch sentence lengths
    int* sent_lens = (int*) calloc(n_batches*batch_size, sizeof(int));
    read_sentence_lens("sentence_lens.txt", sent_lens, n_batches*batch_size);
    
    //Allocate kernels and initilize
    kerns.k = (float *)calloc(kerns.height*kerns.width*kerns.num, sizeof(float));
    init_kerns(kerns.k, kerns.num, kerns.width, kerns.height);
    
    // Test kernel initialization
    if (DEBUG) {
        for (int i=0; i<kerns.num; i++) {
            printf("Kernel: %d\n", i);
            print_mat(&kerns.k[i*kerns.height*kerns.width], kerns.width, kerns.height);
            printf("\n\n");
        }
    }
    
    // Define test idxs
    int test_batch = 9, test_idx = 9;
    
    WORDVECS wordvec;
    OUTPUTS output;

    
    //Select the device you want to run the code.
    hipSetDevice(device_id);
    printf("Using device: %d \n", device_id);
    
    // Allocate GPU WORDVEC, KERNS and OUTPUT. Planning to pass these structs by value.
    WORDVECS d_wordvec;
    KERNS d_kerns;
    OUTPUTS d_output;
    
    // Allocate and Initialize kerns.k on device
    float* d_k;
    hipMalloc((void **) &(d_k), sizeof(float)*kerns.num*kerns.width*kerns.height);
    if (DEBUG) {
        printf("Done allocating d_k \n");
    }
    
    hipMemcpy(d_k, kerns.k, sizeof(float)*kerns.num*kerns.width*kerns.height, hipMemcpyHostToDevice);
    if (DEBUG) {
        printf("Done transfering kerns.k -> d_k \n");
    }
    d_kerns.k = d_k;
    d_kerns.num = kerns.num;
    d_kerns.width = kerns.width;
    d_kerns.height = kerns.height;
    
    // Readback and check if the results are right
    hipMemcpy(kerns.k, d_k, sizeof(float)*kerns.num*kerns.width*kerns.height, hipMemcpyDeviceToHost);
    
    if (DEBUG) {
        printf("GPU kernel values. \n");
        for (int i=0; i<kerns.num; i++) {
            printf("Kernel: %d\n", i);
            print_mat(&kerns.k[i*kerns.height*kerns.width], kerns.width, kerns.height);
            printf("\n\n");
        }
    }
    
    for (int batch=0; batch < n_batches; batch++) {
        wordvec.b_size = batch_size;
        wordvec.dim = dim;
        wordvec.lens = (long*) calloc(batch_size, sizeof(long));
        
        init_lens(wordvec.lens, sent_lens, batch_size, batch);
        
        // Test sentence lens for a given mini-batch
        if (DEBUG && (test_batch == batch)) {
            printf("i=%d, len=%ld \n", 0, wordvec.lens[0]);
            for (int i=1; i < wordvec.b_size; i++) {
                printf("i=%d, len=%ld \n", i, wordvec.lens[i] - wordvec.lens[i-1]);
            }
        }
        
        // Allocate word vectors and initialize
        wordvec.w = (float*) calloc(wordvec.dim*wordvec.lens[batch_size-1], sizeof(float));
        init_wordvecs(wordvec.w, wordvec.dim, wordvec.lens[batch_size-1]);
        
        
        //Testing initialization
        if (DEBUG && (test_batch == batch)) {
            printf("Input: \n");
            if (test_idx == 0) {
                print_mat(&(wordvec.w[0*dim]), wordvec.lens[test_idx], wordvec.dim);
            } else {
                print_mat(&(wordvec.w[wordvec.lens[test_idx-1]*dim]), wordvec.lens[test_idx]-wordvec.lens[test_idx-1], wordvec.dim);
            }
        }
        
        //Allocate and initialize outputs
        output.b_size = batch_size;
        output.dim = kerns.num;
        output.lens = (long*) calloc(batch_size, sizeof(long));
        init_out_lens(&(output.lens), wordvec.lens, batch_size, kerns.width);
        
        // Test output lens for a given mini-batch
        if (DEBUG && test_batch == batch) {
            printf("i=%d, len=%ld, out_len=%ld \n", 0, wordvec.lens[0], output.lens[0]);
            for (int i=1; i < wordvec.b_size; i++) {
                printf("i=%d, len=%ld, out_len=%ld \n", i, wordvec.lens[i] - wordvec.lens[i-1],  output.lens[i]-output.lens[i-1]);
            }
        }
        
        //Allocate outputs
        output.out = (float*) calloc(kerns.num*output.lens[batch_size-1], sizeof(float));
        
        //Run on CPU and test output
        if (DEBUG && test_batch == batch) {
            conv1d(wordvec, kerns, output);
            
            printf("CPU Output: \n");
            if (test_idx == 0) {
                print_mat(&(output.out[0*kerns.num]), output.lens[test_idx], kerns.num);
            } else {
                print_mat(&(output.out[output.lens[test_idx-1]*kerns.num]), output.lens[test_idx]-output
                          .lens[test_idx-1], kerns.num);
            }
            
            memset(output, 0, sizeof(float)*kerns.num*output.lens[batch_size-1]);
        }

        
        // Allocate and initialize wordvecs.w and wordvecs.lens on GPU
        d_wordvec.dim = dim;
        d_wordvec.b_size = batch_size;
        long* d_wlens;
        hipMalloc((void **) &(d_wlens), sizeof(long)*batch_size);
        if (DEBUG) {
            printf("Done allocating d_wlens \n");
        }
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        hipMemcpy(d_wlens, wordvec.lens, sizeof(long)*batch_size, hipMemcpyHostToDevice);
        if (DEBUG) {
            printf("Done transfering wordvecs[batch].lens -> d_wlens \n");
        }
        d_wordvec.lens = d_wlens;
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        float* d_w;
        hipMalloc((void **) &(d_w), sizeof(float)*dim*wordvec.lens[batch_size-1]);
        if (DEBUG) {
            printf("Done allocating d_w \n");
        }
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        hipMemcpy(d_w, wordvec.w, sizeof(float)*dim*wordvec.lens[batch_size-1], hipMemcpyHostToDevice);
        if (DEBUG) {
            printf("Done transfering wordvecs[batch].w -> d_w \n");
        }
        d_wordvec.w = d_w;
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        // Allocate and initialize outputs.out and outputs.lens on GPU
        d_output.dim = kerns.num;
        d_output.b_size = batch_size;
        long* d_olens;
        hipMalloc((void **) &(d_olens), sizeof(long)*batch_size);
        if (DEBUG) {
            printf("Done allocating d_olens \n");
        }
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        hipMemcpy(d_olens, output.lens, sizeof(long)*batch_size, hipMemcpyHostToDevice);
        if (DEBUG) {
            printf("Done transfering wordvecs[batch].lens -> d_olens \n");
        }
        d_output.lens = d_olens;
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        float* d_out;
        hipMalloc((void **) &(d_out), sizeof(float)*kerns.num*output.lens[batch_size-1]);
        if (DEBUG) {
            printf("Done allocating d_out \n");
        }
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        hipMemcpy(d_out, output.out, sizeof(float)*kerns.num*output.lens[batch_size-1], hipMemcpyHostToDevice);
        if (DEBUG) {
            printf("Done transfering outputs[batch].out -> d_out \n");
        }
        d_output.out = d_out;
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        // Launch the kernel
        
        conv1d_kernel<<<batch_size, dim, sizeof(float)*dim>>>(d_wordvec, d_kerns, d_output);
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        if (DEBUG) {
            printf("Done launching the kernel. \n");
        }
        
        // Get output results back
        hipMemcpy(output.out, d_out, sizeof(float)*kerns.num*output.lens[batch_size-1], hipMemcpyDeviceToHost);
        if (DEBUG) {
            printf("Done transfering d_out -> outputs[batch].out \n");
        }
        
        err = hipGetLastError();
        if (err != hipSuccess)
            printf("***ERROR***: %s\n", hipGetErrorString(err));
        
        // Verify GPU Results
        if (DEBUG) {
            if (batch == test_batch) {
                printf("GPU Output: \n");
                if (test_idx == 0) {
                    print_mat(&(output.out[0*kerns.num]), output.lens[test_idx], kerns.num);
                } else {
                    print_mat(&(output.out[output.lens[test_idx-1]*kerns.num]), output.lens[test_idx]-output.lens[test_idx-1], kerns.num);
                }
            }
        }
        
        // Free GPU allocations for mini-batch
        hipFree(d_wlens);
        hipFree(d_w);
        hipFree(d_olens);
        hipFree(d_out);
        
        // Free all allocated resources.
        free(wordvec.w);
        free(wordvec.lens);
        free(output.out);
        free(output.lens);
    }
    
    //Free all GPU allocated resources.
    hipFree(d_k);
    free(kerns.k);
    free(sent_lens);
}